#include "hip/hip_runtime.h"
#include "spMat_mine_gpu.h"
#include <hip/hip_runtime.h>
#include <>
#include <cassert>

const static int blockSize = 256;

__global__ void spmat_mul_vec(vType* d_result,
	const int* d_OuterStarts, const int* d_ColIndices, const vType* d_Values, const vType* d_vec,
	const int rows, const int b_rows)
{
	unsigned int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int offset = threadIdx.y * b_rows;
	if (row_idx >= rows)
	{
		return;
	}

	vType tmp_val = 0;
	for (int ith_elem = d_OuterStarts[row_idx]; ith_elem < d_OuterStarts[row_idx + 1]; ++ith_elem)
	{
		tmp_val += d_Values[ith_elem] * d_vec[d_ColIndices[ith_elem] + offset];
	}
	d_result[row_idx + offset] = tmp_val;
}

// ������������ָ��֮��Ĵ棬�����Ļ����쳣��ſ���Ҳ���Ƚϰ�ȫ
// �����ֱ��c���ֿռ���

void CudaCheck()
{
	if (hipGetLastError() != hipSuccess)
	{
		throw std::runtime_error("Cuda Failed");
	}
}

spMat_mine_gpu::spMat_mine_gpu(const std::vector<std::tuple<int, int, vType>>& data, int rows, int cols)
	:spMat_mine(data, rows, cols)
{
	assign_space_and_cpy_to_gpu();
}

void spMat_mine_gpu::assign(const std::vector<std::tuple<int, int, vType>>& data, int rows, int cols)
{
	spMat_mine::assign(data, rows, cols);
	assign_space_and_cpy_to_gpu();
}

std::vector<vType> spMat_mine_gpu::MatMul(const std::vector<vType>& vec)
{
	assert(vec.size() % Cols == 0);
	int b_cols = vec.size() / Cols;
	vType* d_result, *d_vec;

	hipMalloc(&d_vec, vec.size() * sizeof(vType));
	hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(vType), hipMemcpyHostToDevice);

	hipMalloc(&d_result, vec.size() * sizeof(vType));

	// ���� Rows*b_cols ������
	int numThreads = std::min(blockSize, Rows);
	int numBlocks = (Rows % numThreads != 0) ? (Rows / numThreads + 1) : (Rows / numThreads);

	dim3 grid(numBlocks, 1, 1), block(numThreads, b_cols, 1);
	spmat_mul_vec << <grid, block >> > (d_result, _d_OuterStarts, _d_ColIndices, _d_Values, d_vec, Rows, Cols);

	std::vector<vType> ret(vec.size());
	hipMemcpy(ret.data(), d_result, vec.size() * sizeof(vType), hipMemcpyDeviceToHost);

	hipFree(d_vec);
	hipFree(d_result);
	CudaCheck();
	return ret;
}

spMat_mine_gpu::~spMat_mine_gpu()
{
	free_space_gpu();
}

void spMat_mine_gpu::assign_space_and_cpy_to_gpu()
{
	if (_d_ColIndices || _d_OuterStarts || _d_Values)
	{
		free_space_gpu();
	}
	hipMalloc(&_d_OuterStarts, OuterStarts.size() * sizeof(decltype(OuterStarts)::value_type));
	hipMalloc(&_d_ColIndices, ColIndices.size() * sizeof(decltype(ColIndices)::value_type));
	hipMalloc(&_d_Values, Values.size() * sizeof(decltype(Values)::value_type));

	hipMemcpy(_d_OuterStarts, OuterStarts.data(), OuterStarts.size() * sizeof(decltype(OuterStarts)::value_type), hipMemcpyHostToDevice);
	hipMemcpy(_d_ColIndices, ColIndices.data(), ColIndices.size() * sizeof(decltype(ColIndices)::value_type), hipMemcpyHostToDevice);
	hipMemcpy(_d_Values, Values.data(), Values.size() * sizeof(decltype(Values)::value_type), hipMemcpyHostToDevice);

	CudaCheck();
}
void spMat_mine_gpu::free_space_gpu()
{
	if (_d_ColIndices)
	{
		hipFree(_d_ColIndices);
		_d_ColIndices = nullptr;
	}
	if (_d_OuterStarts)
	{
		hipFree(_d_OuterStarts);
		_d_OuterStarts = nullptr;
	}
	if (_d_Values)
	{
		hipFree(_d_Values);
		_d_Values = nullptr;
	}
	CudaCheck();
}